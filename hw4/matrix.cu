#include "hip/hip_runtime.h"
// Matrix - Vector Multiplication
// Reference: https://github.com/NYU-HPC19/lecture8/blob/master; Professor Stadler, NYU, who previously taught this course

#include <algorithm>
#include <stdio.h>
#include <omp.h>
#include <string>
#include <stdlib.h>
#include <time.h>
#include <iostream>

// void product_cpu_cal2(double* sum_ptr, const double* a, const double* b, long M, long N){
//     double sum = 0;
//     for(long j = 0; j < M; ++j){
//         sum = 0;
//         #pragma omp parallel for schedule(static) reduction(+:sum)
//         for(long i = 0; i < N; ++i){
//             sum += a[j*M+i] * b[i];
//         }
//         sum_ptr[j] = sum;
//     }
// }

void Check_CUDA_Error(const char *message){
  hipError_t error = hipGetLastError();
  if(error!=hipSuccess) {
    fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
    exit(-1);
  }
}

double error_func(double* x, double* y, int size){
    double total_val = 0.0;
    for(int i = 0; i < size; ++i) total_val = max(fabs(x[i] - y[i]), total_val);
    return total_val;
}

// From previous HWs
void MMult(long M, long N, double* A, double* x, double* c){
    #pragma omp parallel for
    for(long i = 0; i < M; ++i){
        for(long j = 0; j < N; ++j){
            c[i] += A[i*M+j] * x[j];
        }
    }
}
#define BLOCK_SIZE 1024

__global__ void reduction_kernel(double* sum, const double* a, long N){
  __shared__ double smem[BLOCK_SIZE];
  int idx = (blockIdx.x) * blockDim.x + threadIdx.x;

  if (idx < N) smem[threadIdx.x] = a[idx];
  else smem[threadIdx.x] = 0;

  __syncthreads();
  if(threadIdx.x < 512) smem[threadIdx.x] += smem[threadIdx.x + 512];
  __syncthreads();
  if(threadIdx.x < 256) smem[threadIdx.x] += smem[threadIdx.x + 256];
  __syncthreads();
  if(threadIdx.x < 128) smem[threadIdx.x] += smem[threadIdx.x + 128];
  __syncthreads();
  if(threadIdx.x <  64) smem[threadIdx.x] += smem[threadIdx.x + 64];
  __syncthreads();
  if(threadIdx.x < 32){
      smem[threadIdx.x] += smem[threadIdx.x + 32];
      __syncwarp();
      smem[threadIdx.x] += smem[threadIdx.x + 16];
      __syncwarp();
      smem[threadIdx.x] += smem[threadIdx.x + 8];
      __syncwarp();
      smem[threadIdx.x] += smem[threadIdx.x + 4];
      __syncwarp();
      smem[threadIdx.x] += smem[threadIdx.x + 2];
      __syncwarp();
      if(threadIdx.x == 0) sum[blockIdx.x] = smem[0] + smem[1];
  }
}

__global__ void reduction_product(double* sum, const double* a, const double* b, long N){
  __shared__ double smem[BLOCK_SIZE];
  int idx = (blockIdx.x) * blockDim.x + threadIdx.x;

  if (idx < N) smem[threadIdx.x] = a[idx] * b[idx];
  else smem[threadIdx.x] = 0;

  __syncthreads();
  if(threadIdx.x < 512) smem[threadIdx.x] += smem[threadIdx.x + 512];
  __syncthreads();
  if(threadIdx.x < 256) smem[threadIdx.x] += smem[threadIdx.x + 256];
  __syncthreads();
  if(threadIdx.x < 128) smem[threadIdx.x] += smem[threadIdx.x + 128];
  __syncthreads();
  if(threadIdx.x < 64) smem[threadIdx.x] += smem[threadIdx.x + 64];
  __syncthreads();
  if(threadIdx.x < 32){
      smem[threadIdx.x] += smem[threadIdx.x + 32];
      __syncwarp();
      smem[threadIdx.x] += smem[threadIdx.x + 16];
      __syncwarp();
      smem[threadIdx.x] += smem[threadIdx.x + 8];
      __syncwarp();
      smem[threadIdx.x] += smem[threadIdx.x + 4];
      __syncwarp();
      smem[threadIdx.x] += smem[threadIdx.x + 2];
      __syncwarp();
      if(threadIdx.x == 0) sum[blockIdx.x] = smem[0] + smem[1];
  }
}

int main(){
    long N = (1UL << 10);
    long M = (1UL << 10);

    double *a, *b;
    hipHostMalloc((void**)&a, N * M * sizeof(double));
    hipHostMalloc((void**)&b, N * sizeof(double));
    #pragma omp parallel for schedule(static)
    for(long i = 0; i < N; ++i){ // add more randomness here
        b[i] = ((double)rand())/RAND_MAX;
    }
    #pragma omp parallel for schedule(static)
    for(long i = 0; i < N*M; ++i){ 
        a[i] = ((double)rand())/RAND_MAX;
    }

    double *sum_ref, *sum;
    sum_ref = (double *)malloc(M*sizeof(double));
    sum = (double *)malloc(M*sizeof(double));
    for(int i = 0; i < M; ++i){
        sum_ref[i] = 0;
        sum[i] = 0;
    }

    double tt = omp_get_wtime();
    MMult(M, N, a, b, sum_ref);
    printf("CPU Bandwidth = %f GB/s\n", 2*M*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);

    double *x_d, *y_d, *z_d;
    hipMalloc(&x_d, N*sizeof(double));
    hipMalloc(&y_d, N*sizeof(double));
    long N_work = 1;
    for (long i = (N+BLOCK_SIZE-1)/(BLOCK_SIZE); i > 1; i = (i+BLOCK_SIZE-1)/(BLOCK_SIZE)) N_work += i;
    hipMalloc(&z_d, N_work*sizeof(double));
    hipMemcpyAsync(y_d, b, N*sizeof(double), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    tt = omp_get_wtime();

    for(int i = 0; i < M; ++i){
        double* sum_d;
        hipMalloc(&sum_d, N_work*sizeof(double));
        hipMemcpyAsync(x_d, &(a[i*N]), N*sizeof(double), hipMemcpyHostToDevice);
        hipDeviceSynchronize();
        long Nb = (N+BLOCK_SIZE-1)/(BLOCK_SIZE);
        reduction_product<<<Nb,BLOCK_SIZE>>>(sum_d, x_d, y_d, N);
        while(Nb > 1){
            long N1 = Nb;
            Nb = (Nb+BLOCK_SIZE-1)/(BLOCK_SIZE);
            reduction_kernel<<<Nb,BLOCK_SIZE>>>(sum_d + N1, sum_d, N1);
            sum_d += N1;
        }
        hipMemcpyAsync(&(sum[i]), sum_d, sizeof(double), hipMemcpyDeviceToHost);
    }
    hipDeviceSynchronize();
    printf("GPU Bandwidth = %f GB/s\n", 2*M*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);
    printf("Error = %f\n", error_func(sum,sum_ref,M));

    hipFree(x_d);
    hipFree(y_d);
    hipFree(z_d);
    free(sum);
    free(sum_ref);

    return 0;
}

